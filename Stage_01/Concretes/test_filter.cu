// test_filter.cu
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void dummy_kernel() {
    printf("CUDA kernel running on Jetson Nano!\n");
}

void test_cuda() {
    dummy_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    std::cout << "CUDA test completed successfully!\n";
}